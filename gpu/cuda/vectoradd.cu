
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


#include <hip/hip_runtime_api.h>

// function to add the elements of two arrays
void add_cpu(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

void on_cpu(){
    std::cout << "Running on cpu." << std::endl;
    
    int N = 1<<20; // 1M elements

    float *x = new float[N];
    float *y = new float[N];

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    add_cpu(N, x, y);

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    delete [] x;
    delete [] y;
}

__global__
void add_gpu(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}
void on_gpu()
{
    hipProfilerStart();

    std::cout << "Running on gpu." << std::endl;
    int N = 1<<20;
    float *x, *y;

    // Allocate Unified Memory  accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add_gpu<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    hipDeviceReset();
    hipProfilerStop();

}

int main(void)
{
    on_cpu();
    on_gpu();
    return 0;
}

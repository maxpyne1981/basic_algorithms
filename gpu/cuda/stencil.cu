
#include <hip/hip_runtime.h>
// only kernel, not fully executable

#define RADIUS 7
#define BLOCK_SIZE 512
__global__ void stencil(int *in, int *out) 
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;
	
    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    // At both end of a block, the sliding window moves beyond the block boundary.
    // E.g, for thread id = 512, we wiil read in[505] and in[1030] into temp.
    if (threadIdx.x < RADIUS) {
       temp[lindex - RADIUS] = in[gindex - RADIUS];
       temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }
 
   __syncthreads();
    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
       result += temp[lindex + offset];

    // Store the result
    out[gindex] = result; 
}
